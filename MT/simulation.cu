#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "fireSim.h"
#include <sys/time.h>
#include "kernel.h"


// const int INF = 999999;

// __device__ int end;

// __global__ void MT(){

//   end = 1;
//   printf("Kernel: %d\n", end);
// }

#define MT 1
#define IMT 0
#define BD 0
const int SIZE = 2048;
int main(){
  // float memTime, calcTime;
  hipError_t devError = hipSetDevice(0);
  // std::cerr << "Error: " << hipGetErrorString(devError) << std::endl;
  // hipDeviceProp_t prop;
  // devError = hipGetDeviceProperties ( &prop,0);
  // cout << "MaxthreadsPerBlock: " << prop.maxThreadsPerBlock << endl;
  // cout << "Name: " << prop.name << endl;
  // cout << "RegPerBlock: " << prop.regsPerBlock << endl;
  // int SIMTYPE = 1;
  int B = 1024;
  int T = 128; 
  for(int S = SIZE; S <= SIZE; S<<=1){
    cout << "Timing: " << S  << "x" << S << "Size" << endl;
      // Declare simulation variables
      // int cell, row, col, nrow, ncol, ncell;
      // char simType[20];
      std::ofstream fout;

      // Initialize simulator
      fireSim sim(S,S);
      struct timeval start, fin;

    sim.init();
    sim.updateSpreadData();

    // Allocate Roth Data for GPU
    float* gpuRoth;
    int* gpuTime;
    int* timeSteppers;
    float* loc_L_n;
    float* loc_burnDist;
    bool* check;
    gpuRoth = (float*)malloc(sim.simDimX*sim.simDimY*3*sizeof(float));
    gpuTime = (int*)malloc(sim.simDimX*sim.simDimY*sizeof(int));
    timeSteppers = (int*)malloc(2*sizeof(int));
    loc_L_n = (float*)malloc(16*sizeof(float));
    check = (bool*)malloc(sim.simDimX*sim.simDimY*sizeof(bool));
    loc_burnDist = (float*)malloc(8*sim.simDimX*sim.simDimY*sizeof(float));


    for(int k = 0, cell = 0, tcell = 0; k < sim.simDimX; k++){
      for(int c = 0; c < sim.simDimY; c++, cell+=3, tcell++){
        // cout << cell << endl;
        gpuRoth[cell + 0] = sim.rothData[k][c].x;
        gpuRoth[cell + 1] = sim.rothData[k][c].y;
        gpuRoth[cell + 2] = sim.rothData[k][c].z;
        gpuTime[tcell] = sim.ignTime[tcell];

        check[tcell] = false;
      }
    }

    // Allocate Time data for GPU 
    // float* timeSteppers = new float[2];
    // cout << "CPU: " << endl;
    for(int i = 0; i < 16; i++){
      loc_L_n[i] = sim.L_n[i];
    }

    timeSteppers[0] = 0;
    timeSteppers[1] = 0;
    // timeSteppers[1] = INF;

    char simType[20];

    sprintf(simType, "../out/MT");

    // sprintf(simType, "../out/GPU_DEBUG");
    // sprintf(simType, "../out/GPU_DEBUG");
   
    // Allocate Cuda Variables
    gettimeofday(&start, NULL);
    int *g_ignTime;
    float *g_rothData;
    int *g_times;
    float *g_L_n;

    hipError_t err = hipMalloc( (void**) &g_ignTime, sim.simDimX*sim.simDimY*sizeof(int));
    err = hipMalloc( (void**) &g_rothData, sim.simDimX*sim.simDimY*3*sizeof(float));
    err = hipMalloc( (void**) &g_times, 2*sizeof(int));
    err = hipMalloc( (void**) &g_L_n, 16*sizeof(float));

    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
      }

    err = hipMemcpy(g_ignTime, gpuTime, sim.simDimX*sim.simDimY*sizeof(int), hipMemcpyHostToDevice);
    err = hipMemcpy(g_rothData, gpuRoth, sim.simDimX*sim.simDimY*3*sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(g_times, timeSteppers, 2*sizeof(int), hipMemcpyHostToDevice);
    err = hipMemcpy(g_L_n, loc_L_n, 16*sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    // Kernel Loop
    int counter = 0;
    // terminate = 0;
    cout << "Kicking off Kernels" << endl;
    typeof(syncCounter) terminate = -1;
//    int B = 1024;
    // int T = 100;
//    int T = sim.simDimX*sim.simDimY / B;
    // int B = S;
    // int T = S;

    //if(T >= 1024){
//      T = S;
    if(S < B)
      B = S;
    if(S < T) 
      T = S; 
      // B = sim.simDimX*sim.simDimY / T;
    //}
    while(terminate <= 0){
    // while(counter < 1969){
      counter++;
      // Do calculations
     MinTime<<<B,T>>>(g_ignTime, g_rothData, 
                           g_times, g_L_n, sim.simDimX*sim.simDimY,
                           sim.simDimX, sim.simDimY);
      // Update Time Kernel 
      timeKernelMT<<<1,1>>>(g_times);

      // hipDeviceSynchronize();
      err = hipMemcpyFromSymbol(&terminate, HIP_SYMBOL(end), sizeof(end), 0, 
                                 hipMemcpyDeviceToHost);
      // err = hipMemcpyFromSymbol(&terminate, HIP_SYMBOL(syncCounter), sizeof(syncCounter), 0, 
      //                            hipMemcpyDeviceToHost);
      if (err != hipSuccess) {
          std::cerr << "Error copying from GPU: " << hipGetErrorString(err) << std::endl;
          exit(1);
      }
      // cout << terminate <<endl;
      // if(terminate < sim.simDimX*sim.simDimY)
      //   terminate = -1;

      if(terminate < 4)
        terminate = -1;
    }
    int finishCount = 0;
    // Catch last corner to terminate simulation
    while(finishCount <= 3){
      counter++;
      finishCount++;
      // Do calculations
      MinTime<<<B,T>>>(g_ignTime, g_rothData, 
                           g_times, g_L_n, sim.simDimX*sim.simDimY,
                           sim.simDimX, sim.simDimY);
      // Update Time Kernel 
      timeKernelMT<<<1,1>>>(g_times);
    }
    terminate = 0;
    // hipEventRecord(end, 0);
    // hipEventSynchronize(end);

    // hipEventElapsedTime( &calcTime, start, end);
    cout << "Simulation Complete" << endl;
    // Copy back to device
    err = hipMemcpy(gpuTime, g_ignTime, sim.simDimX*sim.simDimY*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error copying from GPU: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

      // hipEventRecord(m_end, 0);
      // hipEventSynchronize(m_end);

      // hipEventElapsedTime(&memTime, m_start, m_end);

      gettimeofday(&fin, NULL);

      double t_init = fin.tv_usec + fin.tv_sec * 1000000.0;
      t_init -= start.tv_usec + start.tv_sec * 1000000.0;
      t_init /= 1000000.0;   
      std::cout << "Processing init on " << sim.simDimX << " cells took " << t_init << " seconds" << std::endl;
      
    // hipEventDestroy( start );
    // hipEventDestroy( end );
    // hipEventDestroy( m_start );
    // hipEventDestroy( m_end );

      // Free memory
      hipFree(g_ignTime);
      hipFree(g_rothData);
      hipFree(g_times);
      hipFree(g_L_n);

      // Write data to file
      char threadNum[21];
      sprintf(threadNum, "_%d_%d", sim.simDimX, sim.simDimY);
      char csv[] = ".csv";
      strcat(simType,threadNum);
      strcat(simType,csv);
      fout.open(simType);
      printf("Using %d Blocks and %d Threads with %d Iterations\n", B,T,counter);
      printf("Writing to %s\n", simType);
      for(int i = 0; i < sim.simDimX*sim.simDimY; i++){
        // std::cout << ignTime[i] << " ,";
        if(i %sim.simDimX == 0 && i !=0){
            // std::cout << std::endl;
            fout << '\n';
        }
        // fout << (int)sim.ignTime[i] << " ";
        // fout << (int)ignTimeNew[i] << " ";
        fout << gpuTime[i] / 100<< " ";
      }
      fout.close();
   cout << "-------------" << endl << endl;
   cout << gpuTime[0] << endl;
   }


   return 0;
}
